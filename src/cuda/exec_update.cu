#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <sys/time.h> 
#include <stdio.h>
#include <math.h> 

#include "api/datatype.hpp"

#define maxwalklength 10

__device__ vid_t getSourceId( WalkDataType walk ){
    return (vid_t)( walk >> 40 ) & 0xffffff;
}

__device__ vid_t getCurrentId( WalkDataType walk ){
    return (vid_t)( walk >> 14 ) & 0x3ffffff;
}

__device__ hid_t getHop( WalkDataType walk ){
    return (hid_t)(walk & 0x3fff) ;
}

__device__ WalkDataType encode( vid_t sourceId, vid_t currentId, hid_t hop ){
    assert( hop < 16384 );
    return (( (WalkDataType)sourceId & 0xffffff ) << 40 ) |(( (WalkDataType)currentId & 0x3ffffff ) << 14 ) | ( (WalkDataType)hop & 0x3fff ) ;
}

__device__ int gpu_rand_r(unsigned int *seed){
    unsigned int next = *seed;
    int result;

    next *= 1103515245;
    next += 12345;
    result = (unsigned int) (next / 65536) % 2048;

    next *= 1103515245;
    next += 12345;
    result <<= 10;
    result ^= (unsigned int) (next / 65536) % 1024;

    next *= 1103515245;
    next += 12345;
    result <<= 10;
    result ^= (unsigned int) (next / 65536) % 1024;

    *seed = next;

    return result;
}
 
__global__ void updatebywalk(sid_t exec_interval, vid_t* intervals, eid_t* beg_pos, vid_t* csr, WalkDataType* walks, WalkDataType* pwalks, wid_t* pnwalks, vid_t nverts, wid_t nwalks, sid_t nshards){
    unsigned i = blockDim.x * blockIdx.x + threadIdx.x;
    while( i < nwalks ){
        // if(blockDim.x * blockIdx.x + threadIdx.x == 0) {
            // printf("\nin GPU, pnwalks[0] = %d, ", pnwalks[0] );
            // printf("pnwalks[1] = %d\n ", pnwalks[1] );
            // printf("pwalks[0] = %d\n ", pwalks[0] );
            // printf("nwalks = %d\n ", nwalks );
            WalkDataType nowWalk = walks[i];
            //random walk
            vid_t sourId = getSourceId(nowWalk);
            vid_t dstId = getCurrentId(nowWalk) + intervals[exec_interval];
            hid_t hop = getHop(nowWalk);
            unsigned seed = i+dstId+hop; //+cur_time;
            bool reset = false;
            // printf("%d ， [%d, %d), %d \n", dstId, intervals[exec_interval], intervals[exec_interval+1], hop );
            while (dstId >= intervals[exec_interval] && dstId < intervals[exec_interval+1] && hop < maxwalklength ){
                // std::cout  << " -> " << dstId << " " << getSourceId(nowWalk) << std::endl;
                // updateInfo(sourId, dstId, threadid, hop);
                eid_t outd = beg_pos[dstId+1] - beg_pos[dstId];
                // printf("dstId = %d\n ", dstId );
                // printf("outd = %d\n ", outd );
                if (outd > 0 && ((float)gpu_rand_r(&seed))/RAND_MAX > 0.15 ){
                    eid_t pos = beg_pos[dstId] + ((eid_t)gpu_rand_r(&seed))%outd;
                    dstId = csr[pos];
                    // printf(" pos = %d", pos );
                    // printf(" move to --> %d\n", dstId );
                }else{
                    // printf("%d : Reset!\n", i);
                    reset = true;
                    break;
                }
                hop++;
            }
            // printf("hop = %d, ", hop );
            // printf("maxwalklength = %d\n ", maxwalklength );
            if( hop < maxwalklength && !reset ){
                sid_t p;
                for(p = 0; p < nshards; p++){
                    if(dstId < intervals[p+1]) {
                        // printf("p = %d, ", p );
                        // printf("dstId = %d\n ", dstId );
                        break;
                    }
                }
                // printf("after break : dstId = %d\n ", dstId );
                nowWalk = encode(sourId, dstId-intervals[p], hop);
                // printf("after encode : nowWalk = %d\n ", nowWalk );
                // printf("before : p = %d, ", p );
                // printf("pnwalks[p] = %d\n ", pnwalks[p] );
                wid_t w = p*nwalks + pnwalks[p]++;
                pwalks[w] = nowWalk;
                // printf("after : w = %d, ", w );
                // printf("pnwalks[p] = %d\n ", pnwalks[p] );
                // walk_manager.setMinStep( p, hop );
            }
        // }
        //Next walk
        i += 28*1024;
        // printf("i = %d, ", i );
        // printf("nwalks = %d\n ", nwalks );
    }
}
 
//int exec_update(RandomWalk &userprogram, Vertex *&vertices, WalkManager &walk_manager )
// int main(){
void exec_updates(eid_t *beg_pos, vid_t *csr, sid_t exec_interval, vid_t* intervals, WalkDataType* walks, WalkDataType **&pwalks, wid_t *&pnwalks, vid_t nverts, eid_t nedges, wid_t nwalks, sid_t nshards){
    struct timeval start, end;
    gettimeofday( &start, NULL );

    // printf("in exec_updates : exec_interval = %d, nshards = %d, nverts = %d, , nedges = %d, nwalks = %d\n", exec_interval, nshards, nverts, nedges, nwalks);

    //define the variables used in GPU
    vid_t* d_intervals;
    eid_t* d_beg_pos;
    vid_t* d_csr;
    WalkDataType* d_walks; // walks in current interval copied to GPU
    WalkDataType* d_pwalks; //walks moved to other intervals
    wid_t* d_pnwalks;

    // std::cout << "before malloc device memory" << std::endl;
    //malloc device memory
    hipMalloc((void**)&d_intervals, sizeof(vid_t) * (nshards+1));
    hipMalloc((void**)&d_beg_pos, sizeof(eid_t) * (nverts+1));
    hipMalloc((void**)&d_csr, sizeof(vid_t) * nedges);
    hipMalloc((void**)&d_walks, sizeof(WalkDataType) * nwalks);

    hipMalloc((void**)&d_pnwalks, sizeof(wid_t) * nshards);
    hipMemset(d_pnwalks, 0, sizeof(wid_t) * nshards);

    hipMalloc((void**)&d_pwalks, sizeof(WalkDataType) * nshards * nwalks);
    hipMemset(d_pwalks, 0, sizeof(WalkDataType) * nshards * nwalks);

    // std::cout << "before hipMemcpy" << std::endl;
    hipMemcpy(d_intervals, intervals, sizeof(vid_t)*(nshards+1), hipMemcpyHostToDevice);
    hipMemcpy(d_beg_pos, beg_pos, sizeof(eid_t)*(nverts+1), hipMemcpyHostToDevice);
    hipMemcpy(d_csr, csr, sizeof(vid_t)*nedges, hipMemcpyHostToDevice);
    hipMemcpy(d_walks, walks, sizeof(WalkDataType)*nwalks, hipMemcpyHostToDevice);
    hipMemcpy(d_pnwalks, pnwalks, sizeof(wid_t)*nshards, hipMemcpyHostToDevice);

    // printf("in CPU, pnwalks[0] = %d, ", pnwalks[0] );
    // printf("pnwalks[1] = %d\n ", pnwalks[1] );

    // 定义kernel执行配置，28个block，每个block里面有1024个线程
    dim3 dimGrid(28);
    dim3 dimBlock(1024);

	// hipMemcpy2D(d_pwalks, pitch, pwalks, sizeof(WalkDataType) * nwalks, sizeof(WalkDataType) * nwalks, nshards, hipMemcpyHostToDevice);
    
    //conduct random walk moving
    // std::cout << "before updatebywalk" << std::endl;
    updatebywalk<<<dimGrid, dimBlock>>>(exec_interval, d_intervals, d_beg_pos, d_csr, d_walks, d_pwalks, d_pnwalks, nverts, nwalks, nshards);

	// 将device端数据拷贝到host端返回数据
    std::cout << "before hipMemcpyDeviceToHost;" << std::endl;
    hipMemcpy(pnwalks, d_pnwalks, sizeof(wid_t) * nshards, hipMemcpyDeviceToHost);
    for(sid_t p = 0; p < nshards; p++){
        printf("pnwalks[%d] = %d, \n", p, pnwalks[p]);
        pwalks[p] = (WalkDataType*)malloc(sizeof(WalkDataType) * pnwalks[p]);
        hipMemcpy(pwalks[p], d_pwalks, sizeof(WalkDataType) * pnwalks[p], hipMemcpyDeviceToHost);
    }

    //释放设备内存
    std::cout << "before hipFree(d_intervals);" << std::endl;
    hipFree(d_intervals);
    hipFree(d_beg_pos);
    hipFree(d_csr);
    hipFree(d_walks);
    hipFree(d_pnwalks);
    hipFree(d_pwalks);

    gettimeofday( &end, NULL );
    // int timeuse = 1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
    // printf("total time in exec_update is %d ms\n", timeuse/1000);

}
