#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <sys/time.h> 
#include <stdio.h>
#include <math.h> 

#include "walks/randomwalk.hpp"

#define maxwalklength 6

__device__ vid_t getSourceId( WalkDataType walk ){
    return (vid_t)( walk >> 40 ) & 0xffffff;
}

__device__ vid_t getCurrentId( WalkDataType walk ){
    return (vid_t)( walk >> 14 ) & 0x3ffffff;
}

__device__ unsigned getHop( WalkDataType walk ){
    return (unsigned)(walk & 0x3fff) ;
}

__device__ WalkDataType encode( vid_t sourceId, vid_t currentId, unsigned hop ){
    assert( hop < 16384 );
    return (( (WalkDataType)sourceId & 0xffffff ) << 40 ) |(( (WalkDataType)currentId & 0x3ffffff ) << 14 ) | ( (WalkDataType)hop & 0x3fff ) ;
}

__device__ int gpu_rand_r(unsigned int *seed){
    unsigned int next = *seed;
    int result;

    next *= 1103515245;
    next += 12345;
    result = (unsigned int) (next / 65536) % 2048;

    next *= 1103515245;
    next += 12345;
    result <<= 10;
    result ^= (unsigned int) (next / 65536) % 1024;

    next *= 1103515245;
    next += 12345;
    result <<= 10;
    result ^= (unsigned int) (next / 65536) % 1024;

    *seed = next;

    return result;
}
 
__global__ void updatebywalk(unsigned exec_interval, unsigned* intervals, unsigned* beg_pos, unsigned* csr, WalkDataType* walks, WalkDataType** pwalks, unsigned nvertices, unsigned nwalks, unsigned nshards){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    while( i < nwalks ){
    printf("\nupdatebywalk : %d \n", i );
        WalkDataType nowWalk = walks[i];
        //random walk
        vid_t sourId = getSourceId(nowWalk);
        vid_t dstId = getCurrentId(nowWalk) + intervals[exec_interval];
        unsigned hop = getHop(nowWalk);
        unsigned seed = i+dstId+hop; //+cur_time;
        printf("%d ， [%d, %d], %d \n", dstId, intervals[exec_interval], intervals[exec_interval+1], hop );
        while (dstId >= intervals[exec_interval] && dstId < intervals[exec_interval+1] && hop < maxwalklength ){
            // std::cout  << " -> " << dstId << " " << getSourceId(nowWalk) << std::endl;
            // updateInfo(sourId, dstId, threadid, hop);
            unsigned outd = beg_pos[dstId+1] - beg_pos[dstId];
            printf("%d : outd : %d \n ", i, outd );
            if (outd > 0 ){//&& ((float)gpu_rand_r(&seed))/RAND_MAX > 0.15 ){
                printf("i = %d, dstId = %d , beg_pos[dstId] = %d-- ", i, dstId, beg_pos[dstId] );
                unsigned pos = beg_pos[dstId] + ((unsigned)gpu_rand_r(&seed))%outd;
                dstId = csr[pos];
                printf(" pos = %d move to --> %d\n", pos, dstId );
            }else{
                printf("%d : Reset!\n", i);
                break;
            }
            hop++;
            nowWalk++;
        }
        if( hop < maxwalklength ){
            unsigned p = 0;
            for(; p < nshards; p++){
                if(dstId < intervals[p]) break;
            }
            nowWalk = encode(sourId, dstId-intervals[p], hop);
            // pwalks[p].push_back(nowWalk);
            // walk_manager.setMinStep( p, hop );
        }

        //Next walk
        i += 28*1024;
    }
    
    
}
 
//int exec_update(RandomWalk &userprogram, Vertex *&vertices, WalkManager &walk_manager )
int main(){
    struct timeval start, end;
    gettimeofday( &start, NULL );

    //input data
    unsigned nvertices = 3; // number of vertices in current interval
    unsigned nedges = 7; // number of vertices in current interval
    unsigned nwalks = 5; // number of walks in current interval
    unsigned nshards = 3; // number of shards

    unsigned intervals[] = {0, 3, 6, 10};
    unsigned beg_pos[] = {0, 3, 4, 7}; //interval 0
    unsigned csr[] = {1,3,4,2,0,6,7};
    WalkDataType walks[] = {0,0,2,16384,16385}; // walks in current interval
    WalkDataType** pwalks; // walks in current interval

    //define the variables used in GPU
    unsigned* d_intervals;
    unsigned* d_beg_pos; //interval 0
    unsigned* d_csr;
    WalkDataType* d_walks; // walks in current interval copied to GPU
    WalkDataType** d_pwalks; //walks moved to other intervals

    std::cout << "before malloc device memory" << std::endl;
    //malloc device memory
    hipMalloc((void**)&d_intervals, sizeof(unsigned) * (nshards+1));
    hipMalloc((void**)&d_beg_pos, sizeof(unsigned) * (nvertices+1));
    hipMalloc((void**)&d_csr, sizeof(unsigned) * nedges);
    hipMalloc((void**)&d_walks, sizeof(WalkDataType) * nwalks);

    std::cout << "before malloc pwalks memory" << std::endl;
    pwalks = (WalkDataType**)malloc(sizeof(WalkDataType*) * nshards);
    // hipMalloc((void**)&d_pwalks, sizeof(WalkDataType*) * nshards);
    for(unsigned p = 0; p < nshards; p++){
        pwalks[p] = (WalkDataType*)malloc(sizeof(WalkDataType) * nwalks);
        // hipMalloc((void**)&d_pwalks[p], sizeof(WalkDataType) * nwalks);
    }
    size_t pitch;
    hipMallocPitch((void**)&d_pwalks, &pitch, sizeof(WalkDataType) * nwalks, nshards);

    std::cout << "before hipMemcpy" << std::endl;
    hipMemcpy(d_intervals, intervals, sizeof(unsigned)*(nshards+1), hipMemcpyHostToDevice);
    hipMemcpy(d_beg_pos, beg_pos, sizeof(unsigned)*(nshards+1), hipMemcpyHostToDevice);
    hipMemcpy(d_csr, csr, sizeof(unsigned)*nedges, hipMemcpyHostToDevice);
    hipMemcpy(d_walks, walks, sizeof(WalkDataType)*nwalks, hipMemcpyHostToDevice);

    // 定义kernel执行配置，28个block，每个block里面有1024个线程
    dim3 dimGrid(28);
    dim3 dimBlock(1024);

    std::cout << "before updatebywalk" << std::endl;
    //conduct random walk moving
    updatebywalk <<<dimGrid, dimBlock>>> (0, d_intervals, d_beg_pos, d_csr, d_walks, d_pwalks, nvertices, nwalks, nshards);

    //拷贝计算数据-一级数据指针
    hipMemcpy2D(pwalks, sizeof(WalkDataType) * nwalks, d_pwalks, pitch, sizeof(WalkDataType) * nwalks, nshards, hipMemcpyDeviceToHost);
    // hipMemcpy(pwalks, d_pwalks, sizeof(WalkDataType) * nwalks, hipMemcpyDeviceToHost);

    //释放主机内存
    // free(intervals);
    // free(beg_pos);
    // free(csr);
    // free(walks);
    std::cout << "\nbefore free(pwalks)[0]" << std::endl;
    for(unsigned p = 0; p < nshards; p++)
        free(pwalks[p]);
    // std::cout << "before free(pwalks)" << std::endl;
    // free(pwalks);
    //释放设备内存
    std::cout << "before hipFree(d_intervals);" << std::endl;
    hipFree(d_intervals);
    hipFree(d_beg_pos);
    hipFree(d_csr);
    hipFree(d_walks);
    std::cout << "before hipFree(d_pwalks);" << std::endl;
    hipFree(d_pwalks);
    // for(unsigned p = 0; p < nshards; p++)
    //     hipFree(d_pwalks[p]);

    gettimeofday( &end, NULL );
    int timeuse = 1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
    printf("total time in exec_update is %d ms\n", timeuse/1000);

    return 0;
}
