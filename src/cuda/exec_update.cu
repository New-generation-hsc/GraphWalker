#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <sys/time.h> 
#include <stdio.h>
#include <math.h> 

#include "api/datatype.hpp"
#include "metrics/metrics.hpp"

#define maxwalklength 10
#define walkpitch 1000
#define nthreads 28*1024

__device__ vid_t getSourceId( WalkDataType walk ){
    return (vid_t)( walk >> 40 ) & 0xffffff;
}

__device__ vid_t getCurrentId( WalkDataType walk ){
    return (vid_t)( walk >> 14 ) & 0x3ffffff;
}

__device__ hid_t getHop( WalkDataType walk ){
    return (hid_t)(walk & 0x3fff) ;
}

__device__ WalkDataType encode( vid_t sourceId, vid_t currentId, hid_t hop ){
    assert( hop < 16384 );
    return (( (WalkDataType)sourceId & 0xffffff ) << 40 ) |(( (WalkDataType)currentId & 0x3ffffff ) << 14 ) | ( (WalkDataType)hop & 0x3fff ) ;
}

__device__ int gpu_rand_r(unsigned int *seed){
    unsigned int next = *seed;
    int result;

    next *= 1103515245;
    next += 12345;
    result = (unsigned int) (next / 65536) % 2048;

    next *= 1103515245;
    next += 12345;
    result <<= 10;
    result ^= (unsigned int) (next / 65536) % 1024;

    next *= 1103515245;
    next += 12345;
    result <<= 10;
    result ^= (unsigned int) (next / 65536) % 1024;

    *seed = next;

    return result;
}
 
__global__ void updatebywalk(sid_t exec_interval, vid_t* intervals, eid_t* beg_pos, vid_t* csr, WalkDataType* walks, WalkDataType* tpwalks, wid_t* tpnwalks, vid_t nverts, wid_t nwalks, sid_t nshards){
    sid_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned i = tid;
    while( i < nwalks ){
        WalkDataType nowWalk = walks[i];
        vid_t sourId = getSourceId(nowWalk);
        vid_t dstId = getCurrentId(nowWalk) + intervals[exec_interval];
        hid_t hop = getHop(nowWalk);
        unsigned seed = i+dstId+hop; //+cur_time;
        bool reset = false;
        if (dstId < intervals[exec_interval] || dstId >= intervals[exec_interval+1] || hop >= maxwalklength ){
        }
        while (dstId >= intervals[exec_interval] && dstId < intervals[exec_interval+1] && hop < maxwalklength ){
            vid_t dstIdptr = dstId - intervals[exec_interval];
            eid_t outd = beg_pos[dstIdptr+1] - beg_pos[dstIdptr];
            if (outd > 0 && ((float)gpu_rand_r(&seed))/RAND_MAX > 0.15 ){
                eid_t pos = beg_pos[dstIdptr] + ((eid_t)gpu_rand_r(&seed))%outd;
                dstId = csr[pos];
            }else{
                reset = true;
                break;
            }
            hop++;
        }
        if( hop < maxwalklength && !reset ){
            sid_t p;
            for(p = 0; p < nshards; p++){
                if(dstId < intervals[p+1]) {
                    break;
                }
            }
            nowWalk = encode(sourId, dstId-intervals[p], hop);
            wid_t w = tid*(nshards*walkpitch) + p*walkpitch + tpnwalks[tid*nshards+p];
            if(tpnwalks[tid*nshards+p] < walkpitch-1) tpnwalks[tid*nshards+p]++;
            tpwalks[w] = nowWalk;
            // if(w == walkpitch+2) printf("pwalks[w] = %lld, \n", tpwalks[w]);
        }
        i += nthreads; //Next walk
    }
}
 
//int exec_update(RandomWalk &userprogram, Vertex *&vertices, WalkManager &walk_manager )
// int main(){
void exec_updates(metrics &m, eid_t *beg_pos, vid_t *csr, sid_t exec_interval, vid_t* intervals, WalkDataType* walks, WalkDataType **&pwalks, wid_t *&pnwalks, vid_t nverts, eid_t nedges, wid_t nwalks, sid_t nshards){
    struct timeval start, end;
    gettimeofday( &start, NULL );

    //define the variables used in GPU
    vid_t* d_intervals;
    eid_t* d_beg_pos;
    vid_t* d_csr;
    WalkDataType* d_walks; // walks in current interval copied to GPU
    WalkDataType* d_tpwalks; //walks moved to other intervals
    wid_t* d_tpnwalks;

    wid_t* tpnwalks;

    m.start_time("CPU_GPU_Memcpy");
    // std::cout << "before malloc device memory" << std::endl;
    //malloc device memory and copy data from host to device
    hipMalloc((void**)&d_intervals, sizeof(vid_t) * (nshards+1));
    hipMalloc((void**)&d_beg_pos, sizeof(eid_t) * (nverts+1));
    hipMalloc((void**)&d_csr, sizeof(vid_t) * nedges);
    hipMalloc((void**)&d_walks, sizeof(WalkDataType) * nwalks);

    hipMemcpy(d_intervals, intervals, sizeof(vid_t)*(nshards+1), hipMemcpyHostToDevice);
    hipMemcpy(d_beg_pos, beg_pos, sizeof(eid_t)*(nverts+1), hipMemcpyHostToDevice);
    hipMemcpy(d_csr, csr, sizeof(vid_t)*nedges, hipMemcpyHostToDevice);
    hipMemcpy(d_walks, walks, sizeof(WalkDataType)*nwalks, hipMemcpyHostToDevice);

    //malloc device memory for appended walks
    hipMalloc((void**)&d_tpnwalks, sizeof(wid_t) * nthreads*nshards);
    hipMemset(d_tpnwalks, 0, sizeof(wid_t) * nthreads*nshards);
    tpnwalks = (wid_t*)malloc(sizeof(wid_t) * nthreads*nshards);

    hipMalloc((void**)&d_tpwalks, sizeof(WalkDataType) * nthreads*walkpitch*nshards);
    hipMemset(d_tpwalks, 0, sizeof(WalkDataType) * nthreads*walkpitch*nshards);
    m.stop_time("CPU_GPU_Memcpy");

    // 定义kernel执行配置，28个block，每个block里面有1024个线程
    dim3 dimGrid(28);
    dim3 dimBlock(1024);
    
    // printf("exec_interval = %d, nverts = %d, nedges = %d \n", exec_interval, nverts, nedges);
    m.start_time("exec_updates in GPU");
    updatebywalk<<<dimGrid, dimBlock>>>(exec_interval, d_intervals, d_beg_pos, d_csr, d_walks, d_tpwalks, d_tpnwalks, nverts, nwalks, nshards);
    m.stop_time("exec_updates in GPU");

    m.start_time("CPU_GPU_Memcpy");
    // std::cout << "before hipMemcpyDeviceToHost;" << std::endl;
    hipMemcpy(tpnwalks, d_tpnwalks, sizeof(wid_t) * nthreads*nshards, hipMemcpyDeviceToHost);
    for(sid_t p = 0; p < nshards; p++){
        pnwalks[p] = 0;
        for(sid_t t = 0; t < nthreads; t++){
            if(tpnwalks[t*nshards+p] >= walkpitch) printf("tpnwalks[%d][%d] = %d, pnwalks[%d] = %d, \n", t, p, tpnwalks[t*nshards+p], p, pnwalks[p]);
            pnwalks[p] += tpnwalks[t*nshards+p];
        }
        // printf("pnwalks[%d] = %d, \n", p, pnwalks[p]);
    }
    // 将device端数据拷贝到host端返回数据
    for(sid_t p = 0; p < nshards; p++){
        pwalks[p] = (WalkDataType*)malloc(sizeof(WalkDataType) * pnwalks[p]);
        unsigned off = 0;
        for(sid_t t = 0; t < nthreads; t++){
            hipMemcpy(pwalks[p]+off, d_tpwalks + t*nshards*walkpitch+p*walkpitch, sizeof(WalkDataType) * tpnwalks[t*nshards+p], hipMemcpyDeviceToHost);
            off += tpnwalks[t*nshards+p];
        }
    }
    m.stop_time("CPU_GPU_Memcpy");

    //释放设备内存
    // std::cout << "before hipFree(d_intervals);" << std::endl;
    hipFree(d_intervals);
    hipFree(d_beg_pos);
    hipFree(d_csr);
    hipFree(d_walks);
    hipFree(d_tpwalks);
    hipFree(d_tpnwalks);
    free(tpnwalks);

    gettimeofday( &end, NULL );
    // int timeuse = 1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
    // printf("total time in exec_update is %d ms\n", timeuse/1000);
}
